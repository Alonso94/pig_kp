#include "hip/hip_runtime.h"
# include <torch/types.h>
  
#include <ATen/ATen.h>

# include <hip/hip_runtime.h>
# include <hip/hip_runtime.h>

namespace{
    // the sigmoid function
    template<typename scalar_t>
    __device__ __forceinline__ scalar_t sigmoid(scalar_t z) {
        return 1.0 / (1.0 + exp(-z));
    }
    // the derivative of the sigmoid function
    template<typename scalar_t>
    __device__ __forceinline__ scalar_t d_sigmoid(scalar_t z) {
        return sigmoid(z) * (1 - sigmoid(z));
    }
    // the entropy function
    template<typename scalar_t>
    __device__ __forceinline__ scalar_t entropy(scalar_t z) {
        return -z * log(z+1e-8);
    }
    // the derivative of the entropy function
    template<typename scalar_t>
    __device__ __forceinline__ scalar_t d_entropy(scalar_t z) {
        return -1.0 / log(10.0) * (1.0+log(z+1e-8));
    }
    // the kernel function to compute the histogram
    template<typename scalar_t>
    __device__ __forceinline__ scalar_t kernel(scalar_t d, scalar_t L, scalar_t B) {
        return sigmoid((d+L/2)/B) - sigmoid((d-L/2)/B);
    }
    // the derivative of the kernel function
    template<typename scalar_t>
    __device__ __forceinline__ scalar_t d_kernel(scalar_t d, scalar_t L, scalar_t B) {
        return 1/B * (d_sigmoid((d+L/2)/B) - d_sigmoid((d-L/2)/B));
    }
    // initialize the bins as constant memory using linespace
    __constant__ float bins[255];
    // the dimensions of the input image as a constant memory
    __constant__ int d_SF;
    // the kernel function to compute the histogram
    template<typename scalar_t>
    __global__ void joint_entropy_cuda_forward_kernel(torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> input,
                                                torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> joint_entropy_out,
                                                float L,
                                                float B,
                                                int patch_size
                                                ){
        // the image index
        int n = blockIdx.y;
        // the order of the image
        int sf = blockIdx.z;
        // the index of the first patch
        int p = blockIdx.x;
        // the thread index
        int t = threadIdx.x;
        // first image index
        int n1 = sf;
        // second image index
        int n2 = (sf+1)%d_SF;
        float prob = 0, depth_prob=0;
        #pragma unroll
        for(int i=0;i<patch_size;i++){
            prob += (kernel(input[n][n1][p][i][0]-t,L,B)*(kernel(input[n][n2][p][i][0]-t,L,B)))/patch_size;
            prob += (kernel(input[n][n1][p][i][1]-t,L,B)*(kernel(input[n][n2][p][i][1]-t,L,B)))/patch_size;
            prob += (kernel(input[n][n1][p][i][2]-t,L,B)*(kernel(input[n][n2][p][i][2]-t,L,B)))/patch_size;
            depth_prob += (kernel(input[n][n1][p][i][3]-t,L,B)*(kernel(input[n][n2][p][i][3]-t,L,B)))/patch_size;
        }
        // update the output
        atomicAdd(&joint_entropy_out[n][n2][0][p],entropy(prob));
        atomicAdd(&joint_entropy_out[n][n2][1][p],entropy(depth_prob));
    }
    // the kernel function to compute the histogram
    template<typename scalar_t>
    __global__ void joint_entropy_cuda_backward_kernel(torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> input,
                                                torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> d_joint_entropy,
                                                torch::PackedTensorAccessor32<scalar_t,5,torch::RestrictPtrTraits> grad_out,
                                                float L,
                                                float B,
                                                int patch_size
                                                ){
        // the image index
        int n = blockIdx.y;
        // the order of the image
        int sf = blockIdx.z;
        // the index of the patch
        int p = blockIdx.x;
        // the thread index
        int t = threadIdx.x;
        // first image index
        int n1 = sf;
        // second image index
        int n2 = (sf+1)%d_SF;
        float prob = 0, depth_prob=0;
        #pragma unroll
        for(int i=0;i<patch_size;i++){
            prob += (kernel(input[n][n1][p][i][0]-t,L,B)/patch_size)*(kernel(input[n][n2][p][i][0]-t,L,B)/patch_size);
            prob += (kernel(input[n][n1][p][i][1]-t,L,B)/patch_size)*(kernel(input[n][n2][p][i][1]-t,L,B)/patch_size);
            prob += (kernel(input[n][n1][p][i][2]-t,L,B)/patch_size)*(kernel(input[n][n2][p][i][2]-t,L,B)/patch_size);
            depth_prob += (kernel(input[n][n1][p][i][3]-t,L,B)/patch_size)*(kernel(input[n][n2][p][i][3]-t,L,B)/patch_size);
        }
        // the derivative of the entropy function
        float d_prob1 = d_joint_entropy[n][n1][0][p] * d_entropy(prob);
        float d_prob2 = d_joint_entropy[n][n2][0][p] * d_entropy(prob);
        float d_depth_prob1 = d_joint_entropy[n][n2][1][p] * d_entropy(depth_prob);
        float d_depth_prob2 = d_joint_entropy[n][n1][1][p] * d_entropy(depth_prob);
        float d_prob_n1 = 0, d_prob_n2 = 0, d_depth_prob_n1 = 0, d_depth_prob_n2 = 0;
        // compute the gradient
        #pragma unroll
        for(int i=0;i<patch_size;i++){
            d_prob_n1 += d_kernel(input[n][n1][p][i][0]-t,L,B)/patch_size * (kernel(input[n][n2][p][i][0]-t,L,B)/patch_size);
            d_prob_n1 += d_kernel(input[n][n1][p][i][1]-t,L,B)/patch_size * (kernel(input[n][n2][p][i][1]-t,L,B)/patch_size);
            d_prob_n1 += d_kernel(input[n][n1][p][i][2]-t,L,B)/patch_size * (kernel(input[n][n2][p][i][2]-t,L,B)/patch_size);
            grad_out[n][n1][0][p][i] += d_prob1 * d_prob_n1;
            d_prob_n2 += d_kernel(input[n][n2][p][i][0]-t,L,B)/patch_size * (kernel(input[n][n1][p][i][0]-t,L,B)/patch_size);
            d_prob_n2 += d_kernel(input[n][n2][p][i][1]-t,L,B)/patch_size * (kernel(input[n][n1][p][i][1]-t,L,B)/patch_size);
            d_prob_n2 += d_kernel(input[n][n2][p][i][2]-t,L,B)/patch_size * (kernel(input[n][n1][p][i][2]-t,L,B)/patch_size);
            grad_out[n][n2][0][p][i] += d_prob2 * d_prob_n2;
            d_depth_prob_n1 += d_kernel(input[n][n1][p][i][3]-t,L,B)/patch_size * (kernel(input[n][n2][p][i][3]-t,L,B)/patch_size);
            grad_out[n][n1][1][p][i] += d_depth_prob1 * d_depth_prob_n1;
            d_depth_prob_n2 += d_kernel(input[n][n2][p][i][3]-t,L,B)/patch_size * (kernel(input[n][n1][p][i][3]-t,L,B)/patch_size);
            grad_out[n][n2][1][p][i] += d_depth_prob2 * d_depth_prob_n2;
        }
    }
} // namespace

// the forward pass of the joint_entropy layer
// x: input tensor
// region_size: the size of the region
// bandwidth: the bandwidth of the kernel
torch::Tensor joint_entropy_cuda_forward(torch::Tensor x, float bandwidth){
    // printf("joint_entropy_cuda_forward\n");
    // hipError_t cudaStatus;
	// hipEvent_t start, stop;
	// hipEventCreate(&start);
	// hipEventCreate(&stop);
    // the parameters for the kernel function
    const float L=1.0/255.0;
    const float B=bandwidth;
    // get the shape of the input tensor
    // N x SF x P x R x C
    int N = x.size(0);
    int SF = x.size(1);
    int P = x.size(2);
    int R = x.size(3);
    // move SF to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_SF), &SF, sizeof(int));
    // block size
    dim3 threads(256);
    // grid size
    dim3 grid(P,N,SF);
    // define the output tensor
    // N x SF x 2 x P
    auto joint_entropy_output = torch::zeros({N,SF,2,P}).to(x.device());
    int blockSize;
	int minGridSize;
	// hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, joint_entropy_cuda_forward_kernel<float>); 
	// printf("BlockSize: %d\n", blockSize);
	// printf("MinGridSize: %d\n", minGridSize);

    // printf("input tensor shape: %d %d %d\n",N, SF, P, R);
    // printf("grid: %d %d %d\n",grid.x,grid.y,grid.z);
    // printf("threads: %d %d\n",threads.x,threads.y);
    // hipEventRecord(start,0);
    hipFuncSetAttribute(reinterpret_cast<const void*>(joint_entropy_cuda_forward_kernel<float>),hipFuncAttributeMaxDynamicSharedMemorySize,65536);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(joint_entropy_cuda_forward_kernel<float>),hipFuncCachePreferL1);
    // call the kernel
    AT_DISPATCH_FLOATING_TYPES(x.type(),"entropy_cuda_forward",([&]{
        joint_entropy_cuda_forward_kernel<float><<<grid,threads>>>(
            x.packed_accessor32<float,5,torch::RestrictPtrTraits>(),
            joint_entropy_output.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
                L,  B,  R);
    }));
    hipDeviceSynchronize();

    // hipEventRecord(stop, 0);
	// hipEventSynchronize(stop);
    // // Check for any errors launching the kernel
	// cudaStatus = hipGetLastError();
	// if (cudaStatus != hipSuccess) {
	// 	fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	// }

	// // hipDeviceSynchronize waits for the kernel to finish, and returns
	// // any errors encountered during the launch.
	// cudaStatus = hipDeviceSynchronize();
	// if (cudaStatus != hipSuccess) {
	// 	fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	// }

	// float milliseconds = 0;
	// hipEventElapsedTime(&milliseconds, start, stop);

	// std::cout << "GPU rendering required " << milliseconds/1000.0f << "s." << std::endl;

    // return the output
    return joint_entropy_output;
}

// the backward pass of the joint_entropy layer
// x: input tensor
// region_size: the size of the region
// bandwidth: the bandwidth of the kernel
torch::Tensor joint_entropy_cuda_backward(torch::Tensor x,
                            torch::Tensor d_joint_entropy_out,
                            float bandwidth){
    // printf("joint_entropy_cuda_forward\n");
    // hipError_t cudaStatus;
	// hipEvent_t start, stop;
	// hipEventCreate(&start);
	// hipEventCreate(&stop);
    // the parameters for the kernel function
    const float L=1.0/255.0;
    const float B=bandwidth;
    // get the shape of the input tensor
    // N x SF x P x R x C
    int N = x.size(0);
    int SF = x.size(1);
    int P = x.size(2);
    int R = x.size(3);
    // move SF to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_SF), &SF, sizeof(int));
    // block size
    dim3 threads(256);
    // grid size
    dim3 grid(P,N,SF);
    // define the output tensor (the gradient)
    // N x 2 x P x R 
    auto grad_out = torch::zeros({N,SF,2,P,R}).to(x.device());
    // hipEventRecord(start,0);
    // call the kernel
    AT_DISPATCH_FLOATING_TYPES(x.type(),"entropy_cuda_backward",([&]{
        joint_entropy_cuda_backward_kernel<float><<<grid,threads>>>(
            x.packed_accessor32<float,5,torch::RestrictPtrTraits>(),
            d_joint_entropy_out.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
            grad_out.packed_accessor32<float,5,torch::RestrictPtrTraits>(),
             L,  B,  R);
    }));
    hipDeviceSynchronize();

    // hipEventRecord(stop, 0);
	// hipEventSynchronize(stop);
    // // Check for any errors launching the kernel
	// cudaStatus = hipGetLastError();
	// if (cudaStatus != hipSuccess) {
	// 	fprintf(stderr, "kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	// }

	// // hipDeviceSynchronize waits for the kernel to finish, and returns
	// // any errors encountered during the launch.
	// cudaStatus = hipDeviceSynchronize();
	// if (cudaStatus != hipSuccess) {
	// 	fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	// }

	// float milliseconds = 0;
	// hipEventElapsedTime(&milliseconds, start, stop);

	// std::cout << "GPU rendering required " << milliseconds/1000.0f << "s." << std::endl;

    // return the output
    return grad_out;
}