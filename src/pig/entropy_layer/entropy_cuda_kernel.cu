#include "hip/hip_runtime.h"
# include <torch/types.h>
  
#include <ATen/ATen.h>

# include <hip/hip_runtime.h>
# include <hip/hip_runtime.h>

namespace{
    // the sigmoid function
    template<typename scalar_t>
    __device__ __forceinline__ scalar_t sigmoid(scalar_t z) {
        return 1.0 / (1.0 + exp(-z));
    }
    // the derivative of the sigmoid function
    template<typename scalar_t>
    __device__ __forceinline__ scalar_t d_sigmoid(scalar_t z) {
        return sigmoid(z) * (1 - sigmoid(z));
    }
    // the entropy function
    template<typename scalar_t>
    __device__ __forceinline__ scalar_t entropy(scalar_t z) {
        return -z * log(z+1e-8);
    }
    // the derivative of the entropy function
    template<typename scalar_t>
    __device__ __forceinline__ scalar_t d_entropy(scalar_t z) {
        return -1.0 / log(10.0) * (1.0+log(z+1e-8));
    }
    // the kernel function to compute the histogram
    template<typename scalar_t>
    __device__ __forceinline__ scalar_t kernel(scalar_t d, scalar_t L, scalar_t B) {
        return sigmoid((d+L/2)/B) - sigmoid((d-L/2)/B);
    }
    // the derivative of the kernel function
    template<typename scalar_t>
    __device__ __forceinline__ scalar_t d_kernel(scalar_t d, scalar_t L, scalar_t B) {
        return 1/B * (d_sigmoid((d+L/2)/B) - d_sigmoid((d-L/2)/B));
    }
    // initialize the bins as constant memory using linespace
    __constant__ float bins[255];
    // the dimensions of the input image as a constant memory
    __constant__ int d_C, d_H, d_W;
    // the kernel function to compute the histogram
    template<typename scalar_t>
    __global__ void entropy_cuda_forward_kernel(torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> input,
                                                torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> entropy_out,
                                                float L,
                                                float B,
                                                int region_size
                                                ){
        // the image index
        int n = blockIdx.x;
        // x and y coordinates of the center of the patch
        int x_coord = blockIdx.y;
        int y_coord = blockIdx.z;
        // the channel index
        int c = threadIdx.y;
        // the index of the thread
        int t = threadIdx.x;
        // initialize the histogram as zero
        __shared__ scalar_t hist[3][256];
        if (t<255 and c<d_C){
            hist[c][t]=0;
        }
        __syncthreads();
        // the start and the end, row and column of the patch
        int start_row = max(x_coord-region_size/2,0);
        int start_col = max(y_coord-region_size/2,0);
        int end_row = min(x_coord+region_size/2,d_H-1);
        int end_col = min(y_coord+region_size/2,d_W-1);
        // iterate over the patch and compute the histogram
        for (int i=start_row;i<=end_row;i++)
            for (int j=start_col;j<=end_col;j++){
                // the difference between the bin and the pixel value
                float d = input[n][c][i][j]/255.0-bins[t];
                // the kernel value
                float k_val = kernel(d,L,B);
                // update the histogram using atomic add
                atomicAdd(&hist[c][t],k_val);
            }
        __syncthreads();
        // compute the probability of each pixel value
        float p=hist[c][t]/(region_size*region_size);
        float entropy_val = entropy(p);
        // update the entropy using atomic add
        atomicAdd(&entropy_out[n][c][x_coord][y_coord],entropy_val);
        __syncthreads();
    }
    // the kernel function to compute the histogram
    template<typename scalar_t>
    __global__ void entropy_cuda_backward_kernel(torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> input,
                                                torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> d_entropy,
                                                torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> grad_out,
                                                float L,
                                                float B,
                                                int region_size
                                                ){
        // the image index
        int n = blockIdx.x;
        // x and y coordinates of the center of the patch
        int x = blockIdx.y;
        int y = blockIdx.z;
        // the channel index
        int c = threadIdx.y;
        // the index of the thread
        int t = threadIdx.x;
        // initialize the histogram as zero
        __shared__ scalar_t hist[3][256];
        if (t<255 and c<d_C){
            hist[c][t]=0;
        }
        __syncthreads();
        /// the start and the end, row and column of the patch
        int start_row = max(x-region_size/2,0);
        int start_col = max(y-region_size/2,0);
        int end_row = min(x+region_size/2,d_H-1);
        int end_col = min(y+region_size/2,d_W-1);
        // iterate over the patch and compute the histogram
        for (int i=start_row;i<end_row;i++)
            for (int j=start_col;j<end_col;j++){
                // the difference between the bin and the pixel value
                float d = input[n][c][i][j]/255.0-bins[t];
                // the kernel value
                float k_val = kernel(d,L,B);
                // update the histogram using atomic add
                atomicAdd(&hist[c][t],k_val);
            }
        __syncthreads();
        // compute the entropy
        float p=hist[c][t]/(region_size*region_size);
        // define shared tmp gradient value
        __shared__ float tmp_grad;
        tmp_grad=0;
        // compute the gradient of the entropy w.r.t the probability
        float de_p =  -1.0 / log(10.0) * (1.0+log(p));
        // compute the gradient of the probability w.r.t the histogram
        float dp_hist = 1.0/(region_size*region_size);
        // compute the gradient of the histogram w.r.t the pixel value
        // iterate over the pixels in the patch
        for (int i=start_row;i<end_row;i++)
            for (int j=start_col;j<end_col;j++){
                // the difference between the bin and the pixel value
                float d = input[n][c][i][j]/255.0-bins[t];
                // compute the gradient of the kernel w.r.t the pixel value
                float dh_x = d_kernel(d,L,B);
                // update the gradient using atomic add
                atomicAdd(&tmp_grad,de_p*dp_hist*dh_x);
                // update the gradient of the input image using atomic add
                atomicAdd(&grad_out[n][c][i][j],tmp_grad*d_entropy[n][x][y][c]);
            }
        __syncthreads();
    }
} // namespace

// the forward pass of the entropy layer
// x: input tensor
// region_size: the size of the region
// bandwidth: the bandwidth of the kernel
torch::Tensor entropy_cuda_forward(torch::Tensor x, int region_size, float bandwidth){
    // the parameters for the kernel function
    const float L=1.0/255.0;
    const float B=bandwidth;
    const int R = region_size;
    // get the shape of the input tensor
    // N x SF x C x H x W
    int N = x.size(0);
    int SF = x.size(1);
    int C = x.size(2);
    int H = x.size(3);
    int W = x.size(4);
    // copy C,H,W to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_C),&C,sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_H),&H,sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_W),&W,sizeof(int));
    // reshape the input tensor
    // N*SF x H x W x C
    x = x.reshape({N*SF,C,H,W});
    // ( 255 x C ) threads per block
    const dim3 block_size(255,C);
    // ( N*SF x H x W) blocks per grid
    const dim3 grid_size(N*SF,H,W);
    // define the output tensor (the entropy)
    // N*SF x C x H x W
    auto entropy_out = torch::zeros({N*SF,C,H,W},x.options());
    // initialize the bins
    float h_bins[255];
    for(int i=0;i<255;i++)
        h_bins[i]=i/255.0;
    // copy the bins to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(bins),h_bins,255*sizeof(float),0,hipMemcpyHostToDevice);
    // call the kernel
    AT_DISPATCH_FLOATING_TYPES(x.type(),"entropy_cuda_forward",([&]{
        entropy_cuda_forward_kernel<scalar_t><<<grid_size,block_size>>>(
            x.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
            entropy_out.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
             L,  B,  R );
    }));
    // reshape the output tensor
    // N x SF x C x H x W
    entropy_out = entropy_out.reshape({N,SF,C,H,W});
    // return the output
    return entropy_out;
}

// the backward pass of the entropy layer
// x: input tensor
// region_size: the size of the region
// bandwidth: the bandwidth of the kernel
torch::Tensor entropy_cuda_backward(torch::Tensor x,
                            torch::Tensor d_entropy,
                            int region_size,
                            float bandwidth){
    // the parameters for the kernel function
    const float L=1.0/255.0;
    const float B=bandwidth;
    const int R = region_size;
    // get the shape of the input tensor
    // N x SF x C x H x W
    int N = x.size(0);
    int SF = x.size(1);
    int C = x.size(2);
    int H = x.size(3);
    int W = x.size(4);
    // copy C,H,W to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_C),&C,sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_H),&H,sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_W),&W,sizeof(int));
    // reshape the input tensor
    // N*SF x C x H x W
    auto x_flat = x.reshape({N*SF,C,H,W});
    // ( 255 x C ) threads per block
    const dim3 block_size(255,C);
    // ( N*SF x H x W) blocks
    const dim3 grid_size(N*SF,H,W);
    // define the output tensor (the gradient)
    // N*SF x C x H x W 
    auto grad_out = torch::zeros({N*SF,C,H,W},x.options());
    // ershape the entropy gradient
    // N*SF x C x H x W
    d_entropy = d_entropy.reshape({N*SF,C,H,W});
    // initialize the bins
    float h_bins[255];
    for(int i=0;i<255;i++)
        h_bins[i]=i/255.0;
    // copy the bins to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(bins),h_bins,255*sizeof(float),0,hipMemcpyHostToDevice);
    // call the kernel
    AT_DISPATCH_FLOATING_TYPES(x.type(),"entropy_cuda_backward",([&]{
        entropy_cuda_backward_kernel<scalar_t><<<grid_size,block_size>>>(
            x.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
            d_entropy.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
            grad_out.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
             L,  B,  R);
    }));
    // reshape the output tensor
    // N x SF x C x H x W
    grad_out = grad_out.reshape({N,SF,C,H,W});
    // return the output
    return grad_out;
}